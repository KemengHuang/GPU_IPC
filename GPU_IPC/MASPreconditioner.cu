#include "hip/hip_runtime.h"
//
// MASPreconditioner.cu
// GIPC
//
// created by Kemeng Huang on 2022/12/01
// Copyright (c) 2024 Kemeng Huang. All rights reserved.
//

#include "MASPreconditioner.cuh"
#include "cuda_tools.h"
#include ""

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <vector>
#include <bitset>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "hip/hip_cooperative_groups.h"
using namespace cooperative_groups;
//#include "Eigen/Eigen"
using namespace std;

template <class F>
__device__ __host__ inline F __mm_min(F a, F b)
{
    return a > b ? b : a;
}


template <class F>
__device__ __host__ inline F __mm_max(F a, F b)
{
    return a > b ? a : b;
}

#define BANKSIZE 32
#define DEFAULT_BLOCKSIZE 256
#define DEFAULT_WARPNUM 8
__global__ void _buildCML0(const unsigned int* _neighborStart,
                           unsigned int*       _neighborNum,
                           unsigned int*       _neighborList,
                           unsigned int*       _fineConnectedMsk,
                           int                 vertNum)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= vertNum)
        return;
    int          warpId      = idx / BANKSIZE;
    int          laneId      = idx % BANKSIZE;
    int          numNeighbor = _neighborNum[idx];
    unsigned int connectMsk  = (1U << laneId);
    int          nk          = 0;
    int          startId     = _neighborStart[idx];
    for(int i = 0; i < numNeighbor; i++)
    {
        int vIdConnected     = _neighborList[startId + i];
        int warpIdxConnected = vIdConnected / BANKSIZE;
        if(warpId == warpIdxConnected)
        {
            unsigned int laneIdxConnected = vIdConnected % BANKSIZE;
            connectMsk |= (1U << laneIdxConnected);
        }
        else
        {
            _neighborList[startId + nk] = vIdConnected;
            nk++;
        }
    }
    _neighborNum[idx]      = nk;
    _fineConnectedMsk[idx] = connectMsk;
}

__device__ unsigned int _LanemaskLt(int laneIdx)
{
    return (1U << laneIdx) - 1;
}

__global__ void _preparePrefixSumL0(int* _prefixOriginal, unsigned int* _fineConnectedMsk, int vertNum)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= vertNum)
        return;
    int          warpId      = idx / BANKSIZE;
    int          localWarpId = threadIdx.x / BANKSIZE;
    int          laneId      = idx % BANKSIZE;
    unsigned int connectMsk  = _fineConnectedMsk[idx];
    //unsigned int connectMsk = cacheMask1;
    __shared__ int unsigned cacheMask[DEFAULT_BLOCKSIZE];
    __shared__ int          prefixSum[DEFAULT_WARPNUM];
    if(laneId == 0)
    {
        prefixSum[localWarpId] = 0;
    }
    cacheMask[threadIdx.x] = connectMsk;
    unsigned int visited   = (1U << laneId);
    while(connectMsk != -1)
    {
        unsigned int todo = visited ^ connectMsk;

        if(!todo)
            break;

        unsigned int nextVist = __ffs(todo) - 1;
        visited |= (1U << nextVist);
        connectMsk |= cacheMask[nextVist + localWarpId * BANKSIZE];  //__shfl(cacheMask, nextVist);//?????!!!!!
    }

    _fineConnectedMsk[idx] = connectMsk;

    unsigned int electedPrefix = __popc(connectMsk & _LanemaskLt(laneId));

    if(electedPrefix == 0)
    {
        //prefixSum[warpId]++;
        atomicAdd(prefixSum + localWarpId, 1);
    }

    if(laneId == 0)
    {
        _prefixOriginal[warpId] = prefixSum[localWarpId];
    }
}

__global__ void _buildLevel1(int2*               _levelSize,
                             int*                _coarseSpaceTable,
                             int*                _goingNext,
                             const unsigned int* _fineConnectedMsk,
                             const int*          _prefixSumOriginal,
                             const int*          _prefixOriginal,
                             int                 vertNum)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= vertNum)
        return;
    int warpId      = idx / BANKSIZE;
    int localWarpId = threadIdx.x / BANKSIZE;
    int laneId      = idx % BANKSIZE;

    __shared__ unsigned int electedMask[BANKSIZE];
    __shared__ unsigned int lanePrefix[BANKSIZE * BANKSIZE];
    if(laneId == 0)
    {
        electedMask[localWarpId] = 0;
    }
    if(idx == vertNum - 1)
    {
        _levelSize[1].x = _prefixSumOriginal[warpId] + _prefixOriginal[warpId];
        _levelSize[1].y = (vertNum + BANKSIZE - 1) / BANKSIZE * BANKSIZE;
    }

    unsigned int connMsk = _fineConnectedMsk[idx];

    unsigned int electedPrefix = __popc(connMsk & _LanemaskLt(laneId));

    if(electedPrefix == 0)
    {
        atomicOr(electedMask + localWarpId, (1U << laneId));
    }

    //unsigned int lanePrefix2 = __popc(electedMask[localWarpId] & _LanemaskLt(laneId));
    //lanePrefix2 += _prefixSumOriginal[warpId];

    //unsigned int elected_lane = __ffs(connMsk) - 1;
    //unsigned int theLanePrefix = __shfl(lanePrefix2, elected_lane);

    lanePrefix[threadIdx.x] = __popc(electedMask[localWarpId] & _LanemaskLt(laneId));
    lanePrefix[threadIdx.x] += _prefixSumOriginal[warpId];

    unsigned int elected_lane = __ffs(connMsk) - 1;
    unsigned int theLanePrefix = lanePrefix[elected_lane + BANKSIZE * localWarpId];  //__shfl(lanePrefix, elected_lane);


    _coarseSpaceTable[idx + 0 * vertNum] = theLanePrefix;
    _goingNext[idx] = theLanePrefix + (vertNum + BANKSIZE - 1) / BANKSIZE * BANKSIZE;
}

__global__ void _buildConnectMaskLx(const unsigned int* _neighborStart,
                                    unsigned int*       _neighborNum,
                                    unsigned int*       _neighborList,
                                    int*                _coarseSpaceTable,
                                    unsigned int*       _nextConnectedMsk,
                                    const unsigned int* _fineConnectedMsk,
                                    int                 level,
                                    int                 vertNum)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= vertNum)
        return;
    int warpId      = idx / BANKSIZE;
    int localWarpId = threadIdx.x / BANKSIZE;
    int laneId      = idx % BANKSIZE;

    unsigned int prefixMsk = _fineConnectedMsk[idx];
    unsigned int connMsk   = 0;
    unsigned int coarseIdx = _coarseSpaceTable[(level - 1) * vertNum + idx];
    int          kn        = _neighborNum[idx];
    int          nk        = 0;
    int          startId   = _neighborStart[idx];
    for(int i = 0; i < kn; i++)
    {
        unsigned int connect = _neighborList[startId + i];
        unsigned int coarseConnect = _coarseSpaceTable[(level - 1) * vertNum + connect];

        if(coarseIdx / BANKSIZE == coarseConnect / BANKSIZE)
        {
            unsigned int off = coarseConnect % BANKSIZE;
            connMsk |= (1U << off);
        }
        else
        {
            _neighborList[startId + nk] = connect;
            nk++;
        }
    }

    _neighborNum[idx] = nk;

    __shared__ int cacheMsk[DEFAULT_BLOCKSIZE];
    cacheMsk[threadIdx.x] = 0;

    if(__popc(prefixMsk) == BANKSIZE)
    {
        atomicOr(cacheMsk + localWarpId * BANKSIZE, connMsk);
        connMsk = cacheMsk[localWarpId * BANKSIZE];
        //if (laneId == 0) {
        //	cacheMsk[localWarpId] = 0;
        //}
    }
    else
    {
        unsigned int electedLane = __ffs(prefixMsk) - 1;
        if(connMsk)
        {
            atomicOr(cacheMsk + localWarpId * BANKSIZE + electedLane, connMsk);
        }
        connMsk = cacheMsk[localWarpId * BANKSIZE + electedLane];
    }

    unsigned int electedPrefix = __popc(prefixMsk & _LanemaskLt(laneId));

    if(connMsk && electedPrefix == 0)
    {
        atomicOr(_nextConnectedMsk + coarseIdx, connMsk);
    }
}

__global__ void _nextLevelCluster(unsigned int* _nextConnectedMsk, unsigned int* _nextPrefix, int number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;
    int            warpId      = idx / BANKSIZE;
    int            localWarpId = threadIdx.x / BANKSIZE;
    int            laneId      = idx % BANKSIZE;
    __shared__ int prefixSum[DEFAULT_WARPNUM];
    if(laneId == 0)
    {
        prefixSum[localWarpId] = 0;
    }
    unsigned int connMsk = (1U << laneId);

    connMsk |= _nextConnectedMsk[idx];

    //unsigned int cachedMsk = connMsk;

    __shared__ unsigned int cachedMsk[DEFAULT_BLOCKSIZE];
    cachedMsk[threadIdx.x] = connMsk;
    unsigned int visited   = (1U << laneId);

    while(true)
    {
        unsigned int todo = visited ^ connMsk;

        if(!todo)
            break;

        unsigned int nextVisit = __ffs(todo) - 1;

        visited |= (1U << nextVisit);

        connMsk |= cachedMsk[nextVisit + localWarpId * BANKSIZE];  //__shfl(cachedMsk, nextVisit);
    }

    _nextConnectedMsk[idx] = connMsk;

    unsigned int electedPrefix = __popc(connMsk & _LanemaskLt(laneId));

    if(electedPrefix == 0)
    {
        atomicAdd(prefixSum + localWarpId, 1);
    }

    if(laneId == 0)
        _nextPrefix[warpId] = prefixSum[localWarpId];
}

__global__ void _prefixSumLx(int2*         _levelSize,
                             unsigned int* _nextPrefix,
                             unsigned int* _nextPrefixSum,
                             unsigned int* _nextConnectMsk,
                             int*          _goingNext,
                             int           level,
                             int           levelBegin,
                             int           number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;
    int warpId      = idx / BANKSIZE;
    int localWarpId = threadIdx.x / BANKSIZE;
    int laneId      = idx % BANKSIZE;

    __shared__ unsigned int electedMask[BANKSIZE];
    __shared__ unsigned int lanePrefix[BANKSIZE * BANKSIZE];
    if(laneId == 0)
    {
        electedMask[localWarpId] = 0;
    }

    if(idx == number - 1)
    {
        _levelSize[level + 1].x = _nextPrefixSum[warpId] + _nextPrefix[warpId];
        _levelSize[level + 1].y = levelBegin + (number + BANKSIZE - 1) / BANKSIZE * BANKSIZE;
    }

    unsigned int connMsk = _nextConnectMsk[idx];

    unsigned int electedPrefix = __popc(connMsk & _LanemaskLt(laneId));

    if(electedPrefix == 0)
    {
        atomicOr(electedMask + localWarpId, (1U << laneId));
    }

    lanePrefix[threadIdx.x] = __popc(electedMask[localWarpId] & _LanemaskLt(laneId));
    lanePrefix[threadIdx.x] += _nextPrefixSum[warpId];

    unsigned int elected_lane = __ffs(connMsk) - 1;
    unsigned int theLanePrefix = lanePrefix[elected_lane + BANKSIZE * localWarpId];  //__shfl(lanePrefix, elected_lane);

    _nextConnectMsk[idx] = theLanePrefix;
    _goingNext[idx + levelBegin] =
        theLanePrefix + levelBegin + (number + BANKSIZE - 1) / BANKSIZE * BANKSIZE;
}

__global__ void _computeNextLevel(int*          _coarseSpaceTable,
                                  unsigned int* _nextConnectMsk,
                                  int           level,
                                  int           number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;

    int next = _coarseSpaceTable[(level - 1) * number + idx];
    _coarseSpaceTable[(level)*number + idx] = _nextConnectMsk[next];
}

__global__ void _aggregationKernel(
    int* _denseLevel, int4* _coarseTable, int* _goingNext, int levelNum, int number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;

    int currentId = idx;
    int aggLevel  = levelNum - 1;
    //__shared__ int4 ctable[DEFAULT_BLOCKSIZE];
    int4 ctable;
    for(int l = 0; l < levelNum - 1; l++)
    {
        int next = _goingNext[currentId];

        //int next0 = __shfl(next, 0);
        ////printf("%d   %d   %d    %d\n", next, next0, l,  idx);
        //if (next == next0) {
        //  aggLevel = __mm_min(l, aggLevel);
        //}

        currentId          = next;
        *(&(ctable.x) + l) = next;
    }

    _denseLevel[idx] = aggLevel;

    //printf("%d   %d\n", aggLevel, idx);

    _coarseTable[idx] = ctable;
}


__global__ void _prepareHessian(const __GEIGEN__::Matrix12x12d* Hessians12,
                                const __GEIGEN__::Matrix9x9d*   Hessians9,
                                const __GEIGEN__::Matrix6x6d*   Hessians6,
                                const __GEIGEN__::Matrix3x3d*   Hessians3,
                                const uint4*                    D4Index,
                                const uint3*                    D3Index,
                                const uint2*                    D2Index,
                                const uint32_t*                 D1Index,
                                __GEIGEN__::Matrix96x96T*       P96,
                                int                             numbers4,
                                int                             numbers3,
                                int                             numbers2,
                                int                             numbers1,
                                int*                            _goingNext,
                                int                             levelNum)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= numbers4 + numbers3 + numbers2 + numbers1)
        return;

    if(idx < numbers4)
    {
        int Hid  = idx / 144;
        int qid  = idx % 144;
        int qrid = qid / 12;
        int qcid = qid % 12;

        int vcid = qcid / 3;
        int vrid = qrid / 3;

        auto* nodeInex = &(D4Index[Hid].x);
        int   vertCid  = *(nodeInex + vcid);
        int   vertRid  = *(nodeInex + vrid);

        //int cha = vertCid - vertRid;

        int         roffset = qrid % 3;
        int         coffset = qcid % 3;
        Precision_T Hval    = Hessians12[Hid].m[qrid][qcid];

        int cPid  = vertCid / BANKSIZE;
        int level = 0;
        while(vertCid / BANKSIZE != vertRid / BANKSIZE && level < levelNum)
        {
            level++;
            vertCid = _goingNext[vertCid];
            vertRid = _goingNext[vertRid];
            cPid    = vertCid / BANKSIZE;
        }
        if(level >= levelNum)
        {
            return;
        }
        //int cPid = vertCid / 32;

        atomicAdd(&(P96[cPid].m[(vertRid % BANKSIZE) * 3 + roffset][(vertCid % BANKSIZE) * 3 + coffset]),
                  Hval);

        while(level < levelNum - 1)
        {
            level++;
            vertCid = _goingNext[vertCid];
            vertRid = _goingNext[vertRid];
            cPid    = vertCid / BANKSIZE;
            if(vertCid / BANKSIZE == vertRid / BANKSIZE)
            {

                atomicAdd(&(P96[cPid].m[(vertRid % BANKSIZE) * 3 + roffset][(vertCid % BANKSIZE) * 3 + coffset]),
                          Hval);
            }
        }
    }
    else if(numbers4 <= idx && idx < numbers3 + numbers4)
    {
        idx -= numbers4;
        int Hid = idx / 81;
        int qid = idx % 81;

        int qrid = qid / 9;
        int qcid = qid % 9;

        int vcid = qcid / 3;
        int vrid = qrid / 3;

        auto* nodeInex = &(D3Index[Hid].x);
        int   vertCid  = *(nodeInex + vcid);
        int   vertRid  = *(nodeInex + vrid);
        //int Pid = vertCid / 12;
        //int cha = vertCid - vertRid;

        int roffset = qrid % 3;
        int coffset = qcid % 3;

        Precision_T Hval = Hessians9[Hid].m[qrid][qcid];

        int cPid  = vertCid / BANKSIZE;
        int level = 0;
        while(vertCid / BANKSIZE != vertRid / BANKSIZE && level < levelNum)
        {
            level++;
            vertCid = _goingNext[vertCid];
            vertRid = _goingNext[vertRid];
            cPid    = vertCid / BANKSIZE;
        }
        if(level >= levelNum)
        {
            return;
        }
        atomicAdd(&(P96[cPid].m[(vertRid % BANKSIZE) * 3 + roffset][(vertCid % BANKSIZE) * 3 + coffset]),
                  Hval);

        while(level < levelNum - 1)
        {
            level++;
            vertCid = _goingNext[vertCid];
            vertRid = _goingNext[vertRid];
            cPid    = vertCid / BANKSIZE;
            if(vertCid / BANKSIZE == vertRid / BANKSIZE)
            {
                atomicAdd(&(P96[cPid].m[(vertRid % BANKSIZE) * 3 + roffset][(vertCid % BANKSIZE) * 3 + coffset]),
                          Hval);
            }
        }
    }
    else if(numbers3 + numbers4 <= idx && idx < numbers3 + numbers4 + numbers2)
    {
        idx -= numbers3 + numbers4;
        int Hid = idx / 36;
        int qid = idx % 36;

        int qrid = qid / 6;
        int qcid = qid % 6;

        int vcid = qcid / 3;
        int vrid = qrid / 3;

        auto* nodeInex = &(D2Index[Hid].x);

        int vertCid = *(nodeInex + vcid);
        int vertRid = *(nodeInex + vrid);
        //int Pid = vertCid / 12;
        int cha = vertCid - vertRid;

        int roffset = qrid % 3;
        int coffset = qcid % 3;

        Precision_T Hval = Hessians6[Hid].m[qrid][qcid];

        int cPid  = vertCid / BANKSIZE;
        int level = 0;
        while(vertCid / BANKSIZE != vertRid / BANKSIZE && level < levelNum)
        {
            level++;
            vertCid = _goingNext[vertCid];
            vertRid = _goingNext[vertRid];
            cPid    = vertCid / BANKSIZE;
        }
        if(level >= levelNum)
        {
            return;
        }
        atomicAdd(&(P96[cPid].m[(vertRid % BANKSIZE) * 3 + roffset][(vertCid % BANKSIZE) * 3 + coffset]),
                  Hval);

        while(level < levelNum - 1)
        {
            level++;
            vertCid = _goingNext[vertCid];
            vertRid = _goingNext[vertRid];
            cPid    = vertCid / BANKSIZE;
            if(vertCid / BANKSIZE == vertRid / BANKSIZE)
            {

                atomicAdd(&(P96[cPid].m[(vertRid % BANKSIZE) * 3 + roffset][(vertCid % BANKSIZE) * 3 + coffset]),
                          Hval);
            }
        }
    }
    else
    {
        idx -= numbers2 + numbers3 + numbers4;
        int Hid = idx / 9;
        int qid = idx % 9;

        int qrid = qid / 3;
        int qcid = qid % 3;

        int nodeIndex = D1Index[Hid];

        Precision_T Hval = Hessians3[Hid].m[qrid][qcid];

        int cPid  = nodeIndex / BANKSIZE;
        int Pod   = nodeIndex % BANKSIZE;
        int level = 0;


        atomicAdd(&(P96[cPid].m[Pod * 3 + qrid][Pod * 3 + qcid]), Hval);

        while(level < levelNum - 1)
        {
            level++;
            nodeIndex = _goingNext[nodeIndex];
            Pod       = nodeIndex % BANKSIZE;
            cPid      = nodeIndex / BANKSIZE;
            atomicAdd(&(P96[cPid].m[Pod * 3 + qrid][Pod * 3 + qcid]), Hval);
        }
    }
}

__global__ void __setMassMat_P96(const double*             _masses,
                                 const int*                _goingNext,
                                 __GEIGEN__::Matrix96x96T* _Mat96,
                                 int                       levelNum,
                                 int                       number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;
    int warpId = idx / BANKSIZE;
    int laneId = idx % BANKSIZE;

    Precision_T mass = _masses[idx];

    int Pid = idx / BANKSIZE;
    int Pod = idx % BANKSIZE;

    _Mat96[Pid].m[Pod * 3][Pod * 3]         = mass;
    _Mat96[Pid].m[Pod * 3 + 1][Pod * 3 + 1] = mass;
    _Mat96[Pid].m[Pod * 3 + 2][Pod * 3 + 2] = mass;

    int level = 0;

    while(level < levelNum - 1)
    {
        level++;
        idx = _goingNext[idx];
        Pid = idx / BANKSIZE;
        Pod = idx % BANKSIZE;
        atomicAdd(&(_Mat96[Pid].m[Pod * 3][Pod * 3]), mass);
        atomicAdd(&(_Mat96[Pid].m[Pod * 3 + 1][Pod * 3 + 1]), mass);
        atomicAdd(&(_Mat96[Pid].m[Pod * 3 + 2][Pod * 3 + 2]), mass);
    }
}


__global__ void __inverse2_P96x96(__GEIGEN__::Matrix96x96T*  PMas,
                                  __GEIGEN__::MasMatrixSymf* invP96,
                                  int                        numbers)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= numbers)
        return;

    int matId = idx / (BANKSIZE * 3);
    int i     = idx % (BANKSIZE * 3);
    //int localMatId = threadIdx.x / 96;
    int                    block_matId = threadIdx.x / (BANKSIZE * 3);
    __shared__ Precision_T colm[32 / BANKSIZE][BANKSIZE * 3];
    //invPMas[matId].m[j][i] = 1;
    if(PMas[matId].m[i][i] == 0)
    {
        PMas[matId].m[i][i] = 1;
    }

    __syncthreads();
    __threadfence();

    int         j = 0;
    Precision_T rt;

    while(j < (BANKSIZE * 3))
    {
        __syncthreads();
        __threadfence();

        rt = PMas[matId].m[j][j];

        colm[block_matId][i] = PMas[matId].m[i][j];

        __syncthreads();
        __threadfence();
        if(i == j)
        {

            PMas[matId].m[i][j] = 1;
        }
        else
        {
            PMas[matId].m[i][j] = 0;
        }
        __syncthreads();
        __threadfence();

        PMas[matId].m[j][i] /= rt;

        __syncthreads();
        __threadfence();
        for(int k = 0; k < (BANKSIZE * 3); k++)
        {
            if(k != j)
            {
                Precision_T rate = -colm[block_matId][k];
                __syncthreads();
                __threadfence();

                PMas[matId].m[k][i] += rate * PMas[matId].m[j][i];
            }
        }

        j++;
    }
    __syncthreads();
    __threadfence();
    if(i % 3 < 2)
        PMas[matId].m[i + 1][i] = PMas[matId].m[i][i + 1];
    else
        PMas[matId].m[i][i - 2] = PMas[matId].m[i - 2][i];
    __syncthreads();
    __threadfence();


    for(int j = 0; j < (BANKSIZE * 3); j++)
    {
        //PMas[matId].m[j][i] = sPMas[block_matId][j][i];
        int rowId = j / 3;
        int colId = i / 3;
        int index = 0;
        if(colId >= rowId)
        {
            index = BANKSIZE * rowId - rowId * (rowId + 1) / 2 + colId;
            invP96[matId].M[index].m[j % 3][i % 3] = PMas[matId].m[j][i];
        }
    }
}

__global__ void __inverse3_P96x96(__GEIGEN__::Matrix96x96T*  P96,
                                  __GEIGEN__::Matrix96x96MT* invP96,
                                  int                        numbers)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= numbers)
        return;

    int matId = idx / 96;
    int i     = idx % 96;
    //int localMatId = threadIdx.x / 96;

    for(int j = 0; j < 96; j++)
    {
        if(i == j)
        {
            invP96[matId].m[j][i] = 1;
            if(P96[matId].m[j][i] == 0)
            {
                P96[matId].m[j][i] = 1;
            }
        }
        else
        {
            invP96[matId].m[j][i] = 0;
        }
    }
    __syncthreads();
    __threadfence();
    int         j  = 0;
    Precision_T rt = P96[matId].m[0][0];
    __syncthreads();
    __threadfence();
    while(/*loopId[localMatId]*/ j < 96)
    {
        if(i <= j)
            invP96[matId].m[j][i] /= rt;
        if(i > j)
            P96[matId].m[j][i] /= rt;

        __syncthreads();
        __threadfence();
        for(int k = 0; k < 96; k++)
        {
            if(k != j)
            {
                Precision_T rate = -P96[matId].m[k][j];
                __syncthreads();
                __threadfence();
                if(i <= j)
                    invP96[matId].m[k][i] += rate * invP96[matId].m[j][i];
                if(i > j)
                    P96[matId].m[k][i] += rate * P96[matId].m[j][i];
            }
        }

        __syncthreads();
        __threadfence();
        j++;
        rt = P96[matId].m[j][j];
    }
}


//__global__ void __inverse2_P96x96(__GEIGEN__::Matrix96x96d* P96, __GEIGEN__::Matrix96x96T* invP96, int numbers) {
//  int idx = blockIdx.x * blockDim.x + threadIdx.x;
//  if (idx >= numbers) return;
//
//  int matId = idx / 96;
//  int i = idx % 96;
//  //int localMatId = threadIdx.x / 96;
//
//  for (int j = 0; j < 96; j++)
//  {
//      if (i == j) {
//          invP96[matId].m[j][i] = 1;
//          if (P96[matId].m[j][i] == 0) {
//              P96[matId].m[j][i] = 1;
//          }
//      }
//      else {
//          invP96[matId].m[j][i] = 0;
//      }
//  }
//  __syncthreads();
//  //__shared__ int loopId[3];
//  //__shared__ double tempRate[3];
//
//  //if (i == 0) {
//  //  loopId[localMatId] = 0;
//  //  tempRate[localMatId] = P96[matId].m[0][0];
//  //}
//  int j = 0;
//  Precision_T rt = P96[matId].m[0][0];
//  __syncthreads();
//  while (/*loopId[localMatId]*/j < 96) {
//
//      //const int j = loopId[localMatId];
//      //const double rt = tempRate;//tempRate[localMatId];
//      if (i >= j) {
//          P96[matId].m[j][i] /= rt;
//      }
//      if (i <= j) {
//          invP96[matId].m[j][i] /= rt;
//      }
//      __syncthreads();
//      Precision_T rate = -P96[matId].m[i][j];
//      for (int k = 0; k < 96; k++) {
//          if (i != j) {
//
//              //__syncthreads();
//              if (k <= i) {
//                  invP96[matId].m[i][k] += rate * invP96[matId].m[j][k];
//              }
//              if (k >= j) {
//                  P96[matId].m[i][k] += rate * P96[matId].m[j][k];
//              }
//          }
//      }
//
//      __syncthreads();
//      //if (i == 0) {
//      //  loopId[localMatId]++;
//      //  tempRate[localMatId] = P96[matId].m[j + 1][j + 1];
//      //}
//      j++;
//      rt = P96[matId].m[j][j];
//      //__syncthreads();
//  }
//}



__global__ void __buildMultiLevelR_optimized(const double3* _R,
                                             Precision_T3*  _multiLR,
                                             int*           _goingNext,
                                             unsigned int*  _fineConnectMsk,
                                             int            levelNum,
                                             int            numbers)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= numbers)
        return;

    Precision_T3 r;
    r.x = _R[idx].x;
    r.y = _R[idx].y;
    r.z = _R[idx].z;

    int laneId      = threadIdx.x % BANKSIZE;
    int localWarpId = threadIdx.x / BANKSIZE;
    int level       = 0;
    _multiLR[idx]   = r;

    __shared__ double c_sumResidual[DEFAULT_BLOCKSIZE * 3];

    unsigned int connectMsk = _fineConnectMsk[idx];
    if(__popc(connectMsk) == BANKSIZE)
    {
        for(int iter = 1; iter < BANKSIZE; iter <<= 1)
        {
            r.x += __shfl_down(r.x, iter);
            r.y += __shfl_down(r.y, iter);
            r.z += __shfl_down(r.z, iter);
        }
        //int level = 0;

        if(laneId == 0)
        {
            while(level < levelNum - 1)
            {
                level++;
                idx = _goingNext[idx];
                atomicAdd((&((_multiLR + idx)->x)), r.x);
                atomicAdd((&((_multiLR + idx)->x) + 1), r.y);
                atomicAdd((&((_multiLR + idx)->x) + 2), r.z);
            }
        }
        return;
    }
    else
    {
        int elected_lane = __ffs(connectMsk) - 1;

        c_sumResidual[threadIdx.x]                         = 0;
        c_sumResidual[threadIdx.x + DEFAULT_BLOCKSIZE]     = 0;
        c_sumResidual[threadIdx.x + 2 * DEFAULT_BLOCKSIZE] = 0;
        atomicAdd(c_sumResidual + localWarpId * BANKSIZE + elected_lane, r.x);
        atomicAdd(c_sumResidual + localWarpId * BANKSIZE + elected_lane + DEFAULT_BLOCKSIZE,
                  r.y);
        atomicAdd(c_sumResidual + localWarpId * BANKSIZE + elected_lane + 2 * DEFAULT_BLOCKSIZE,
                  r.z);

        unsigned int electedPrefix = __popc(connectMsk & _LanemaskLt(laneId));
        if(electedPrefix == 0)
        {
            while(level < levelNum - 1)
            {
                level++;
                idx = _goingNext[idx];
                atomicAdd((&((_multiLR + idx)->x)), c_sumResidual[threadIdx.x]);
                atomicAdd((&((_multiLR + idx)->x) + 1),
                          c_sumResidual[threadIdx.x + DEFAULT_BLOCKSIZE]);
                atomicAdd((&((_multiLR + idx)->x) + 2),
                          c_sumResidual[threadIdx.x + DEFAULT_BLOCKSIZE * 2]);
            }
        }
    }
}

__global__ void __buildMultiLevelR(
    const double3* _R, Precision_T3* _multiLR, int* _goingNext, int levelNum, int numbers)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= numbers)
        return;

    Precision_T3 r;
    r.x = _R[idx].x;
    r.y = _R[idx].y;
    r.z = _R[idx].z;

    int level     = 0;
    _multiLR[idx] = r;
    while(level < levelNum - 1)
    {
        level++;
        idx = _goingNext[idx];
        atomicAdd((&((_multiLR + idx)->x)), r.x);
        atomicAdd((&((_multiLR + idx)->x) + 1), r.y);
        atomicAdd((&((_multiLR + idx)->x) + 2), r.z);
    }
}

__global__ void __collectFinalZ(double3*            _Z,
                                const Precision_T3* d_multiLevelZ,
                                const int4*         _coarseTable,
                                int                 levelnum,
                                int                 number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;

    Precision_T3 cz;  // = d_multiLevelZ[idx];
    cz.x          = d_multiLevelZ[idx].x;
    cz.y          = d_multiLevelZ[idx].y;
    cz.z          = d_multiLevelZ[idx].z;
    int4 table    = _coarseTable[idx];
    int* tablePtr = &(table.x);
    for(int i = 1; i < __mm_min(levelnum, 4); i++)
    {
        int now = *(tablePtr + i - 1);
        cz.x += d_multiLevelZ[now].x;
        cz.y += d_multiLevelZ[now].y;
        cz.z += d_multiLevelZ[now].z;
    }

    _Z[idx].x = cz.x;
    _Z[idx].y = cz.y;
    _Z[idx].z = cz.z;
}


__global__ void _schwarzLocalXSym3(const __GEIGEN__::MasMatrixSymf* Pred,
                                   const Precision_T3*              mR,
                                   Precision_T3*                    mZ,
                                   int                              number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;

    int hessianSize = (BANKSIZE * 3) * (BANKSIZE);

    int Hid  = idx / hessianSize;
    int MRid = (idx % hessianSize) / (BANKSIZE);
    int MCid = (idx % hessianSize) % (BANKSIZE);

    int vrid = Hid * BANKSIZE + MRid / 3;
    int vcid = Hid * BANKSIZE + MCid;

    int r3id = MRid % 3;

    int          lvrid = vrid % BANKSIZE;
    int          lvcid = vcid % BANKSIZE;
    Precision_TM rdata = 0;

    __shared__ Precision_T3 smR[BANKSIZE];

    if(threadIdx.x < BANKSIZE)
    {
        smR[threadIdx.x] = mR[vcid];
    }
    __syncthreads();

    if(lvcid >= lvrid)
    {
        int index = BANKSIZE * lvrid - lvrid * (lvrid + 1) / 2 + lvcid;
        rdata     = Pred[Hid].M[index].m[r3id][0] * smR[lvcid].x
                + Pred[Hid].M[index].m[r3id][1] * smR[lvcid].y
                + Pred[Hid].M[index].m[r3id][2] * smR[lvcid].z;
    }
    else
    {
        int index = BANKSIZE * lvcid - lvcid * (lvcid + 1) / 2 + lvrid;
        rdata     = Pred[Hid].M[index].m[0][r3id] * smR[lvcid].x
                + Pred[Hid].M[index].m[1][r3id] * smR[lvcid].y
                + Pred[Hid].M[index].m[2][r3id] * smR[lvcid].z;
    }
    //__syncthreads();
    int  warpId    = threadIdx.x & 0x1f;
    int  landidx   = threadIdx.x % BANKSIZE;
    bool bBoundary = (landidx == 0) || (warpId == 0);

    unsigned int mark     = __ballot(bBoundary);  // a bit-mask
    mark                  = __brev(mark);
    unsigned int interval = __mm_min(__clz(mark << (warpId + 1)), 31 - warpId);

    int maxSize = __mm_min(32, BANKSIZE);
    for(int iter = 1; iter < maxSize; iter <<= 1)
    {
        Precision_TM tmpx = __shfl_down(rdata, iter);
        if(interval >= iter)
        {

            rdata += tmpx;
        }
    }

    if(bBoundary)
    {
        atomicAdd((&(mZ[vrid].x) + MRid % 3), rdata);
    }
}

__global__ void _buildCollisionConnection(unsigned int*     _pConnect,
                                          const int*        _pCoarseSpaceTable,
                                          const const int4* _collisionPair,
                                          int               level,
                                          int               vertNum,
                                          int               number)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= number)
        return;
    int4 MMCVIDI              = _collisionPair[idx];
    int* collitionPairStartId = &(MMCVIDI.x);
    if(MMCVIDI.x >= 0)
    {
        if(MMCVIDI.w < 0)
        {
            MMCVIDI.w = -MMCVIDI.w - 1;
        }
        int cpVertNum = 4;
        int cpVid[4];
        if(_pCoarseSpaceTable)
        {
            for(int i = 0; i < 4; i++)
                cpVid[i] = _pCoarseSpaceTable[collitionPairStartId[i] + (level - 1) * vertNum];
        }
        else
        {
            for(int i = 0; i < 4; i++)
                cpVid[i] = collitionPairStartId[i];
        }

        unsigned int connMsk[4] = {0};

        for(int i = 0; i < 4; i++)
        {
            for(int j = i + 1; j < 4; j++)
            {
                unsigned int myId = cpVid[i];
                unsigned int otId = cpVid[j];

                if(myId == otId)
                {
                    continue;
                }
                if(myId / BANKSIZE == otId / BANKSIZE)
                {
                    connMsk[i] |= (1U << (otId % BANKSIZE));
                    connMsk[j] |= (1U << (myId % BANKSIZE));
                }
            }
        }

        for(int i = 0; i < 4; i++)
            atomicOr(_pConnect + cpVid[i], connMsk[i]);
    }
    else
    {
        int v0I   = -MMCVIDI.x - 1;
        MMCVIDI.x = v0I;
        if(MMCVIDI.z < 0)
        {
            if(MMCVIDI.y < 0)
            {
                MMCVIDI.y = -MMCVIDI.y - 1;
                MMCVIDI.z = -MMCVIDI.z - 1;
                MMCVIDI.w = -MMCVIDI.w - 1;

                int cpVertNum = 4;
                int cpVid[4];
                if(_pCoarseSpaceTable)
                {
                    for(int i = 0; i < 4; i++)
                        cpVid[i] =
                            _pCoarseSpaceTable[collitionPairStartId[i] + (level - 1) * vertNum];
                }
                else
                {
                    for(int i = 0; i < 4; i++)
                        cpVid[i] = collitionPairStartId[i];
                }

                unsigned int connMsk[4] = {0};

                for(int i = 0; i < 4; i++)
                {
                    for(int j = i + 1; j < 4; j++)
                    {
                        unsigned int myId = cpVid[i];
                        unsigned int otId = cpVid[j];

                        if(myId == otId)
                        {
                            continue;
                        }
                        if(myId / BANKSIZE == otId / BANKSIZE)
                        {
                            connMsk[i] |= (1U << (otId % BANKSIZE));
                            connMsk[j] |= (1U << (myId % BANKSIZE));
                        }
                    }
                }

                for(int i = 0; i < 4; i++)
                    atomicOr(_pConnect + cpVid[i], connMsk[i]);
            }
            else
            {
                int cpVertNum = 2;
                int cpVid[2];
                if(_pCoarseSpaceTable)
                {
                    for(int i = 0; i < 2; i++)
                        cpVid[i] =
                            _pCoarseSpaceTable[collitionPairStartId[i] + (level - 1) * vertNum];
                }
                else
                {
                    for(int i = 0; i < 2; i++)
                        cpVid[i] = collitionPairStartId[i];
                }

                unsigned int connMsk[2] = {0};

                for(int i = 0; i < 2; i++)
                {
                    for(int j = i + 1; j < 2; j++)
                    {
                        unsigned int myId = cpVid[i];
                        unsigned int otId = cpVid[j];

                        if(myId == otId)
                        {
                            continue;
                        }
                        if(myId / BANKSIZE == otId / BANKSIZE)
                        {
                            connMsk[i] |= (1U << (otId % BANKSIZE));
                            connMsk[j] |= (1U << (myId % BANKSIZE));
                        }
                    }
                }

                for(int i = 0; i < 2; i++)
                    atomicOr(_pConnect + cpVid[i], connMsk[i]);
            }
        }
        else if(MMCVIDI.w < 0)
        {
            if(MMCVIDI.y < 0)
            {
                MMCVIDI.y = -MMCVIDI.y - 1;
                MMCVIDI.w = -MMCVIDI.w - 1;

                int cpVertNum = 4;
                int cpVid[4];
                if(_pCoarseSpaceTable)
                {
                    for(int i = 0; i < 4; i++)
                        cpVid[i] =
                            _pCoarseSpaceTable[collitionPairStartId[i] + (level - 1) * vertNum];
                }
                else
                {
                    for(int i = 0; i < 4; i++)
                        cpVid[i] = collitionPairStartId[i];
                }

                unsigned int connMsk[4] = {0};

                for(int i = 0; i < 4; i++)
                {
                    for(int j = i + 1; j < 4; j++)
                    {
                        unsigned int myId = cpVid[i];
                        unsigned int otId = cpVid[j];

                        if(myId == otId)
                        {
                            continue;
                        }
                        if(myId / BANKSIZE == otId / BANKSIZE)
                        {
                            connMsk[i] |= (1U << (otId % BANKSIZE));
                            connMsk[j] |= (1U << (myId % BANKSIZE));
                        }
                    }
                }

                for(int i = 0; i < 4; i++)
                    atomicOr(_pConnect + cpVid[i], connMsk[i]);
            }
            else
            {
                int cpVertNum = 3;
                int cpVid[3];
                if(_pCoarseSpaceTable)
                {
                    for(int i = 0; i < 3; i++)
                        cpVid[i] =
                            _pCoarseSpaceTable[collitionPairStartId[i] + (level - 1) * vertNum];
                }
                else
                {
                    for(int i = 0; i < 3; i++)
                        cpVid[i] = collitionPairStartId[i];
                }

                unsigned int connMsk[3] = {0};

                for(int i = 0; i < 3; i++)
                {
                    for(int j = i + 1; j < 3; j++)
                    {
                        unsigned int myId = cpVid[i];
                        unsigned int otId = cpVid[j];

                        if(myId == otId)
                        {
                            continue;
                        }
                        if(myId / BANKSIZE == otId / BANKSIZE)
                        {
                            connMsk[i] |= (1U << (otId % BANKSIZE));
                            connMsk[j] |= (1U << (myId % BANKSIZE));
                        }
                    }
                }

                for(int i = 0; i < 3; i++)
                    atomicOr(_pConnect + cpVid[i], connMsk[i]);
            }
        }
        else
        {
            int cpVertNum = 4;
            int cpVid[4];
            if(_pCoarseSpaceTable)
            {
                for(int i = 0; i < 4; i++)
                    cpVid[i] =
                        _pCoarseSpaceTable[collitionPairStartId[i] + (level - 1) * vertNum];
            }
            else
            {
                for(int i = 0; i < 4; i++)
                    cpVid[i] = collitionPairStartId[i];
            }

            unsigned int connMsk[4] = {0};

            for(int i = 0; i < 4; i++)
            {
                for(int j = i + 1; j < 4; j++)
                {
                    unsigned int myId = cpVid[i];
                    unsigned int otId = cpVid[j];

                    if(myId == otId)
                    {
                        continue;
                    }
                    if(myId / BANKSIZE == otId / BANKSIZE)
                    {
                        connMsk[i] |= (1U << (otId % BANKSIZE));
                        connMsk[j] |= (1U << (myId % BANKSIZE));
                    }
                }
            }

            for(int i = 0; i < 4; i++)
                atomicOr(_pConnect + cpVid[i], connMsk[i]);
        }
    }
}


void MASPreconditioner::BuildConnectMaskL0()
{
    int number    = totalNodes;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    _buildCML0<<<numBlocks, blockSize>>>(
        d_neighborStart, d_neighborNum, d_neighborList, d_fineConnectMask, number);
}

void MASPreconditioner::PreparePrefixSumL0()
{
    int number    = totalNodes;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    _preparePrefixSumL0<<<numBlocks, blockSize>>>(d_prefixOriginal, d_fineConnectMask, number);
}

void MASPreconditioner::BuildLevel1()
{
    int number    = totalNodes;
    int blockSize = BANKSIZE * BANKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;
    //exclusive(d_prefixOriginal, d_prefixSumOriginal); wait to do;
    int warpNum = (number + 31) / 32;
    thrust::exclusive_scan(thrust::device_ptr<int>(d_prefixOriginal),
                           thrust::device_ptr<int>(d_prefixOriginal) + warpNum,
                           thrust::device_ptr<int>(d_prefixSumOriginal));
    _buildLevel1<<<numBlocks, blockSize>>>(d_levelSize,
                                           d_coarseSpaceTables,
                                           d_goingNext,
                                           d_fineConnectMask,
                                           d_prefixSumOriginal,
                                           d_prefixOriginal,
                                           number);
}

void MASPreconditioner::BuildConnectMaskLx(int level)
{
    int number    = totalNodes;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;
    _buildConnectMaskLx<<<numBlocks, blockSize>>>(d_neighborStart,
                                                  d_neighborNum,
                                                  d_neighborList,
                                                  d_coarseSpaceTables,
                                                  d_nextConnectMask,
                                                  d_fineConnectMask,
                                                  level,
                                                  number);
}

void MASPreconditioner::NextLevelCluster(int level)
{
    int number    = h_clevelSize.x;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;
    _nextLevelCluster<<<numBlocks, blockSize>>>(d_nextConnectMask, d_nextPrefix, number);
}

void MASPreconditioner::ComputeNextLevel(int level)
{
    int number    = totalNodes;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;
    _computeNextLevel<<<numBlocks, blockSize>>>(
        d_coarseSpaceTables, d_nextConnectMask, level, number);
}

void MASPreconditioner::PrefixSumLx(int level)
{
    int number     = h_clevelSize.x;
    int levelBegin = h_clevelSize.y;
    int blockSize  = BANKSIZE * BANKSIZE;
    int numBlocks  = (number + blockSize - 1) / blockSize;

    int warpNum = (number + 31) / 32;
    thrust::exclusive_scan(thrust::device_ptr<unsigned int>(d_nextPrefix),
                           thrust::device_ptr<unsigned int>(d_nextPrefix) + warpNum,
                           thrust::device_ptr<unsigned int>(d_nextPrefixSum));

    _prefixSumLx<<<numBlocks, blockSize>>>(
        d_levelSize, d_nextPrefix, d_nextPrefixSum, d_nextConnectMask, d_goingNext, level, levelBegin, number);
}

void MASPreconditioner::AggregationKernel()
{
    int number    = totalNodes;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;
    _aggregationKernel<<<numBlocks, blockSize>>>(
        d_denseLevel, d_coarseTable, d_goingNext, levelnum, number);
}


void MASPreconditioner::computeNumLevels(int vertNum)
{
    int totalSz = 0;
    int nLevel  = 1;
    int levelSz = (vertNum + BANKSIZE - 1) / BANKSIZE * BANKSIZE;
    totalSz += levelSz;

    while(levelSz > BANKSIZE)
    {
        levelSz /= BANKSIZE;

        nLevel++;
        levelSz = (levelSz + BANKSIZE - 1) / BANKSIZE * BANKSIZE;
        totalSz += levelSz;
    }

    levelnum = nLevel;
    printf("level num:  %d\n", levelnum);
    //totalSize = totalSz * SizeRatio;
    totalNodes = vertNum;
}

void MASPreconditioner::BuildCollisionConnection(unsigned int* connectionMsk,
                                                 int*          coarseTableSpace,
                                                 int           level,
                                                 int           cpNum)
{
    int number    = cpNum;
    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    _buildCollisionConnection<<<numBlocks, blockSize>>>(
        connectionMsk, coarseTableSpace, _collisonPairs, level, totalNodes, number);
}

int MASPreconditioner::ReorderRealtime(int cpNum)
{
    CUDA_SAFE_CALL(hipMemset(d_levelSize, 0, levelnum * sizeof(int2)));


    BuildConnectMaskL0();


    //CUDA_SAFE_CALL(hipDeviceSynchronize());
    if(cpNum)
        BuildCollisionConnection(d_fineConnectMask, nullptr, -1, cpNum);
    //CUDA_SAFE_CALL(hipDeviceSynchronize());
    PreparePrefixSumL0();

    //vector<unsigned int> h_fineCMsk(totalSize);
    //CUDA_SAFE_CALL(hipMemcpy(h_fineCMsk.data(), d_prefixOriginal, totalNodes * sizeof(unsigned int), hipMemcpyDeviceToHost));

    //for (int i = 0; i < totalNodes; i++) {
    //  /*char s[40];
    //  itoa(h_fineCMsk[i], s, 2);
    //  printf("%s\n", s);*/
    //  //cout << bitset<sizeof(h_fineCMsk[i]) * 8>(h_fineCMsk[i]) << endl;
    //  cout << h_fineCMsk[i] << endl;
    //}

    //CUDA_SAFE_CALL(hipDeviceSynchronize());
    BuildLevel1();

    //CUDA_SAFE_CALL(hipDeviceSynchronize());
    for(int level = 1; level < levelnum; level++)
    {
        CUDA_SAFE_CALL(hipMemset(d_nextConnectMask, 0, totalNodes * sizeof(int)));

        BuildConnectMaskLx(level);
        //CUDA_SAFE_CALL(hipDeviceSynchronize());
        if(cpNum)
            BuildCollisionConnection(d_nextConnectMask, d_coarseSpaceTables, level, cpNum);


        CUDA_SAFE_CALL(hipMemcpy(&h_clevelSize, d_levelSize + level, sizeof(int2), hipMemcpyDeviceToHost));

        //cout << "hello:    " << h_clevelSize.x << endl;

        NextLevelCluster(level);


        //vector<unsigned int> h_fineCMsk(totalSize);
        //CUDA_SAFE_CALL(hipMemcpy(h_fineCMsk.data(), d_nextPrefix, totalNodes * sizeof(unsigned int), hipMemcpyDeviceToHost));

        //for (int i = 0; i < totalNodes; i++) {
        //  /*char s[40];
        //  itoa(h_fineCMsk[i], s, 2);
        //  printf("%s\n", s);*/
        //  //cout << bitset<sizeof(h_fineCMsk[i]) * 8>(h_fineCMsk[i]) << endl;
        //  cout << h_fineCMsk[i] << endl;
        //}


        //CUDA_SAFE_CALL(hipDeviceSynchronize());
        PrefixSumLx(level);
        //CUDA_SAFE_CALL(hipDeviceSynchronize());
        ComputeNextLevel(level);
        //CUDA_SAFE_CALL(hipDeviceSynchronize());
    }

    CUDA_SAFE_CALL(hipMemcpy(&h_clevelSize, d_levelSize + levelnum, sizeof(int2), hipMemcpyDeviceToHost));

    totalNumberClusters = h_clevelSize.y;

    AggregationKernel();
    //CUDA_SAFE_CALL(hipDeviceSynchronize());

    return totalNumberClusters;

    //vector<unsigned int> h_fineCMsk(totalNumberClusters);
    //CUDA_SAFE_CALL(hipMemcpy(h_fineCMsk.data(), d_goingNext, totalNumberClusters * sizeof(unsigned int), hipMemcpyDeviceToHost));


    //for (int i = 0; i < totalNumberClusters; i++) {
    //  /*char s[40];
    //  itoa(h_fineCMsk[i], s, 2);
    //  printf("%s\n", s);*/
    //  //cout << bitset<sizeof(h_fineCMsk[i]) * 8>(h_fineCMsk[i]) << endl;
    //  cout << i << "    " << h_fineCMsk[i] << endl;
    //}
}

//#include <fstream>

void MASPreconditioner::PrepareHessian(const BHessian& BH, const double* masses)
{
    //hipEvent_t start, end0, end1, end2;
    //hipEventCreate(&start);
    //hipEventCreate(&end0);
    //hipEventCreate(&end1);
    //hipEventCreate(&end2);


    int number = totalNodes;

    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    //cout << totalSize / 32 << endl;
    //hipEventRecord(start);
    __setMassMat_P96<<<numBlocks, blockSize>>>(masses, d_goingNext, d_Mat96, levelnum, totalNodes);

    //hipEventRecord(end0);


    number = BH.DNum[3] * 144 + BH.DNum[2] * 81 + BH.DNum[1] * 36 + BH.DNum[0] * 9;
    numBlocks = (number + blockSize - 1) / blockSize;

    _prepareHessian<<<numBlocks, blockSize>>>(BH.H12x12,
                                              BH.H9x9,
                                              BH.H6x6,
                                              BH.H3x3,
                                              BH.D4Index,
                                              BH.D3Index,
                                              BH.D2Index,
                                              BH.D1Index,
                                              d_Mat96,
                                              BH.DNum[3] * 144,
                                              BH.DNum[2] * 81,
                                              BH.DNum[1] * 36,
                                              BH.DNum[0] * 9,
                                              d_goingNext,
                                              levelnum);

    //hipEventRecord(end1);

    blockSize = 96;
    number    = totalNumberClusters * 3;
    numBlocks = (number + blockSize - 1) / blockSize;
    __inverse2_P96x96<<<numBlocks, blockSize>>>(d_Mat96, d_inverseMat96, number);

    //hipEventRecord(end2);

    //CUDA_SAFE_CALL(hipDeviceSynchronize());

    //float time0, time1, time2, time3, time4;
    //hipEventElapsedTime(&time0, start, end0);
    //hipEventElapsedTime(&time1, end0, end1);
    //hipEventElapsedTime(&time2, end1, end2);

    //printf("\n\ntime0 = %f,  time1 = %f,  time1 = %f\n\n", time0, time1, time2);

    //(hipEventDestroy(start));
    //(hipEventDestroy(end0));
    //(hipEventDestroy(end1));
    //(hipEventDestroy(end2));
}

void MASPreconditioner::BuildMultiLevelR(const double3* R)
{
    int number = totalNodes;

    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    //__buildMultiLevelR << <numBlocks, blockSize >> > (R, d_multiLevelR, d_goingNext, levelnum, number);
    __buildMultiLevelR_optimized<<<numBlocks, blockSize>>>(
        R, d_multiLevelR, d_goingNext, d_fineConnectMask, levelnum, number);
    //vector<double3> h_r(totalSize);
    //CUDA_SAFE_CALL(hipMemcpy(h_r.data(), R, totalNodes * sizeof(double3), hipMemcpyDeviceToHost));

    //for (int i = 0; i < totalSize; i++) {

    //  cout << h_r[i].x << " " << h_r[i].y << " " << h_r[i].z << endl;
    //  //cout << h_fineCMsk[i] << endl;
    //}
}

void MASPreconditioner::SchwarzLocalXSym()
{
    int number    = totalNumberClusters * BANKSIZE * 3;
    int blockSize = BANKSIZE * BANKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    //_schwarzLocalXSym1<<<numBlocks, blockSize>>>(d_MatMas, d_multiLevelR, d_multiLevelZ, number);
    _schwarzLocalXSym3<<<numBlocks, blockSize>>>(d_inverseMat96, d_multiLevelR, d_multiLevelZ, number);
}

void MASPreconditioner::CollectFinalZ(double3* Z)
{
    int number = totalNodes;

    int blockSize = DEFAULT_BLOCKSIZE;
    int numBlocks = (number + blockSize - 1) / blockSize;

    __collectFinalZ<<<numBlocks, blockSize>>>(Z, d_multiLevelZ, d_coarseTable, levelnum, number);

}

void MASPreconditioner::setPreconditioner(const BHessian& BH, const double* masses, int cpNum)
{

    CUDA_SAFE_CALL(hipMemcpy(d_neighborList,
                              d_neighborListInit,
                              neighborListSize * sizeof(unsigned int),
                              hipMemcpyDeviceToDevice));
    //CUDA_SAFE_CALL(hipMemcpy(ipc.pcg_data.MP.d_neighborStart, tetMesh.neighborStart.data(), ipc.vertexNum * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_neighborNum,
                              d_neighborNumInit,
                              totalNodes * sizeof(unsigned int),
                              hipMemcpyDeviceToDevice));


    //hipEventRecord(start);

    ReorderRealtime(cpNum);

    //hipEventRecord(end0);

    CUDA_SAFE_CALL(hipMemset(
        d_Mat96, 0, totalNumberClusters / BANKSIZE * sizeof(__GEIGEN__::Matrix96x96T)));

    PrepareHessian(BH, masses);

    //hipEventRecord(end1);
}

void MASPreconditioner::preconditioning(const double3* R, double3* Z)
{
    CUDA_SAFE_CALL(hipMemset(d_multiLevelR + totalNodes,
                              0,
                              (totalNumberClusters - totalNodes) * sizeof(Precision_T3)));
    CUDA_SAFE_CALL(hipMemset(d_multiLevelZ, 0, (totalNumberClusters) * sizeof(Precision_T3)));

    //hipEvent_t start, end0, end1, end2;
    //hipEventCreate(&start);
    //hipEventCreate(&end0);
    //hipEventCreate(&end1);
    //hipEventCreate(&end2);

    //hipEventRecord(start);
    BuildMultiLevelR(R);
    //hipEventRecord(end0);
    SchwarzLocalXSym();
    //hipEventRecord(end1);
    CollectFinalZ(Z);
    //hipEventRecord(end2);

    //CUDA_SAFE_CALL(hipDeviceSynchronize());

    //float time0, time1, time2, time3, time4;
    //hipEventElapsedTime(&time0, start, end0);
    //hipEventElapsedTime(&time1, end0, end1);
    //hipEventElapsedTime(&time2, end1, end2);

    //printf("\n\npreconditioning  time0 = %f,  time1 = %f,  time1 = %f\n\n", time0, time1, time2);

    //(hipEventDestroy(start));
    //(hipEventDestroy(end0));
    //(hipEventDestroy(end1));
    //(hipEventDestroy(end2));
}

void MASPreconditioner::initPreconditioner(int vertNum, int totalNeighborNum, int4* m_collisonPairs)
{
    //bankSize = 32;
    computeNumLevels(vertNum);
    _collisonPairs = m_collisonPairs;

    CUDA_SAFE_CALL(hipMalloc((void**)&d_denseLevel, vertNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_coarseTable, vertNum * sizeof(int4)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_coarseSpaceTables,
                              vertNum * levelnum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_levelSize, (levelnum + 1) * sizeof(int2)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_goingNext,
                              vertNum * levelnum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_prefixOriginal, vertNum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nextPrefix, vertNum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nextPrefixSum, vertNum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_prefixSumOriginal, vertNum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_fineConnectMask, vertNum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nextConnectMask, vertNum * sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborList, totalNeighborNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborStart, vertNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborStartTemp, vertNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborNum, vertNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborListInit, totalNeighborNum * sizeof(int)));
    //CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborStart, vertNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborNumInit, vertNum * sizeof(int)));

    int totalCluster = ReorderRealtime(0) * 1.05;

    CUDA_SAFE_CALL(hipMalloc((void**)&d_Mat96,
                              totalCluster / BANKSIZE * sizeof(__GEIGEN__::Matrix96x96T)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_inverseMat96,
                              totalCluster / BANKSIZE * sizeof(__GEIGEN__::MasMatrixSymf)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_multiLevelR, totalCluster * sizeof(Precision_T3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_multiLevelZ, totalCluster * sizeof(Precision_T3)));
}

void MASPreconditioner::FreeMAS()
{

    CUDA_SAFE_CALL(hipFree(d_denseLevel));
    CUDA_SAFE_CALL(hipFree(d_coarseSpaceTables));
    CUDA_SAFE_CALL(hipFree(d_levelSize));
    CUDA_SAFE_CALL(hipFree(d_goingNext));
    CUDA_SAFE_CALL(hipFree(d_prefixOriginal));
    CUDA_SAFE_CALL(hipFree(d_nextPrefix));
    CUDA_SAFE_CALL(hipFree(d_nextPrefixSum));
    CUDA_SAFE_CALL(hipFree(d_prefixSumOriginal));
    CUDA_SAFE_CALL(hipFree(d_fineConnectMask));
    CUDA_SAFE_CALL(hipFree(d_nextConnectMask));
    CUDA_SAFE_CALL(hipFree(d_neighborList));
    CUDA_SAFE_CALL(hipFree(d_neighborListInit));
    CUDA_SAFE_CALL(hipFree(d_neighborStart));
    CUDA_SAFE_CALL(hipFree(d_neighborStartTemp));
    CUDA_SAFE_CALL(hipFree(d_neighborNum));
    CUDA_SAFE_CALL(hipFree(d_neighborNumInit));
    CUDA_SAFE_CALL(hipFree(d_Mat96));
    CUDA_SAFE_CALL(hipFree(d_inverseMat96));
    CUDA_SAFE_CALL(hipFree(d_multiLevelR));
    CUDA_SAFE_CALL(hipFree(d_multiLevelZ));
}
